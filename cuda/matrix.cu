#include "hip/hip_runtime.h"
#include "matrix.hpp"
//#include <iostream>
//#include <fstream>

using namespace std;
 
Matrix::Matrix(const size_t rows, const size_t cols) {
  rows_ = rows;
  cols_ = cols;
  arr_ = new double[rows*cols];
};

Matrix::Matrix(const Matrix& rmatr) {
    rows_ = rmatr.get_rows();
    cols_ = rmatr.get_cols();
    arr_ = new double[rows_ * cols_];
    for (size_t i = 0; i < rows_; i++) { 
        for (size_t j = 0; j < cols_; j++) {
            arr_[i * cols_ + j] = rmatr.get_arr()[i*cols_ + j];
        }        
    }

}

 
size_t Matrix::get_rows() const {
    return rows_;
}

 
size_t Matrix::get_cols() const {
    return cols_;
}

const double* Matrix::get_arr() const{
    return arr_;
}

 
void Matrix::fill(const double& value) {
  //cout << "IN FILL" << value << endl;
  for (size_t i = 0; i < rows_; i++) {
    for (size_t j = 0; j < cols_; j++) {
      arr_[i * cols_ + j] = value;
      //cout << arr_[i][j] << ' ';
    };
    //cout << endl;
  };
};

 
void Matrix::swap_rows(const size_t first, const size_t second) {
  /*vector<double> temp = arr_[first];
  arr_[first] = arr_[second];
  arr_[second] = temp;*/
    for (size_t i = 0; i < cols_; i++) {
      double temp = arr_[first*cols_ + i];
      arr_[first*cols_ + i] = arr_[second * cols_ + i];
      arr_[second*cols_ + i] = temp;
  }
};

 
void Matrix::set_diag_to_one(const size_t row_index) {
  if (rows_ != cols_) {
    cout << "rows != cols in set_diag_to_one" << endl;
    throw ("rows != cols in set_diag_to_one\n");
  }
  double temp = arr_[row_index*cols_ + row_index];
  for (size_t i = 0; i < cols_; i++) {
    arr_[row_index*cols_ + i] = arr_[row_index*cols_ + i] / temp;
  }
};

 
void Matrix::set_diag_to_one_r(const size_t row_index) {
  if (rows_ != cols_) {
    cout << "rows != cols in set_diag_to_one_r" << endl;
    throw ("rows != cols in set_diag_to_one_r\n");
  }
  for (size_t i = row_index + 1; i < cols_; i++) {
    arr_[row_index*cols_ + i] = arr_[row_index * cols_ + i] / arr_[row_index * cols_ + row_index];
  }
  arr_[row_index * cols_ + row_index] = 1.;
};


 
void Matrix::plus_row(const size_t mod_row_index, const size_t plus_row_index, const double coeff) {
  for (size_t i = 0; i < cols_; i++) {
    arr_[mod_row_index * cols_ + i] += arr_[plus_row_index * cols_ + i] * coeff;
  }
};


 
void Matrix::print() const {
  cout << "IN PRINT" << endl;
  cout << "rows_: " << rows_ << " cols_: " << cols_ << endl;
  for (size_t i = 0; i < rows_; i++) {
    cout << i << " | ";
    for (size_t j = 0; j < cols_; j++) {
      cout << arr_[i * cols_ + j] << ' ';
    }
    cout << endl;
  }
}

 
void Matrix::to_file(ofstream& file) const {
  //cout << "IN_FILE" << endl;
  for (size_t i = 0; i < rows_; i++) {
    file << arr_[i * cols_ + 0];
    for (size_t j = 1; j < cols_; j++) {
      //cout << i << ' ' << j << endl;
      //cout << arr_[i][j] << ' ';
      file << ',' << arr_[i * cols_ + j];
    }
    //cout << endl;
    file << endl;
  }
};

 
void Matrix::from_file(ifstream& file) {
  //cout << "FROM_FILE" << endl;
  for (size_t i = 0; i < rows_; i++) {
    //file >> arr_[i][0];
    for (size_t j = 0; j < cols_; j++) {
      file >>  arr_[i * cols_ + j];
    }
  }
}

 
double* Matrix::operator[](const size_t index) {
  return arr_ + index*cols_;
};


Matrix::~Matrix() {
  delete[] arr_;
}


__global__ void addKernel(double* c, const double* a, const double* b, size_t size) {
    size_t idx = threadIdx.x + blockIdx.x *blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void fillVecKernel(double* vec, size_t size, double val) {
    size_t idx = threadIdx.x + blockIdx.x *blockDim.x;
    if (idx < size) {
        vec[idx] = val;
    }
}


__global__ void calculateMCoeffKernel(double* l_matr, size_t size, size_t i, size_t j, double* mcoeff) {
    *mcoeff = l_matr[i * size + j] / l_matr[j * size + j];
}

__global__ void plusRowKernel(double* l_matr, double* r_vec, size_t size,
                              size_t mr_idx, size_t pr_idx, double* mcoeff) {
    size_t idx = threadIdx.x + blockIdx.x *blockDim.x;
    if (idx < size) {
        //l_matr[mr_idx * size + idx] -= l_matr[pr_idx * size + idx] * l_matr[pr_idx * size + pr_idx];
        l_matr[mr_idx * size + idx] -= l_matr[pr_idx * size + idx] * (*mcoeff);
    }
}

__global__ void diagElemsToVecKernel(double* l_matr, double* diag_vec, size_t size) {
    size_t idx = threadIdx.x + blockIdx.x *blockDim.x;
    if (idx < size) {
        diag_vec[idx] = l_matr[idx*size + idx];
    }
}

__global__ void setDiagToOneKernel(double* l_matr, double* r_vec, double* diag_vec, size_t size,
                                   size_t mr_idx) {
    size_t idx = threadIdx.x + blockIdx.x *blockDim.x;
    if (idx < size) {
        l_matr[mr_idx * size + idx] /= diag_vec[mr_idx];
    }
}

__global__ void rvecDiagDivKernel(double* r_vec, double* diag_vec, double size) {
    size_t idx = threadIdx.x + blockIdx.x *blockDim.x;
    if (idx < size) {
        r_vec[idx] /= diag_vec[idx];
    }
}


vector<double>& gauss_cuda(const Matrix& l_matr, const vector<double>& r_vect) {
    size_t size = r_vect.size();
    if (l_matr.get_cols() != l_matr.get_rows()) {
        cout << "not square matrix in gauss" << endl;
        throw ("not square matrix in gauss");
    }
    if (size != l_matr.get_rows()) {
        cout << "l_matr size != r_vect size in gauss" << endl;
        throw ("l_matr size != r_vect size in gauss");
    }
    // Делаем копию вектора правой части в виде массива
    double* r_vec = new double[size];
    for (size_t i = 0; i < size; i++) {
        r_vec[i] = r_vect[i];
    }


    dim3 N_threads1(8);
    dim3 N_blocks1(size / 8 + 1);
    dim3 N_threads_once(1);
    dim3 N_blocks_once(1);
    // Выделяем память на девайсе
    double* l_matr_dev;
    double* r_vec_dev;
    double* ret_vec_dev;
    size_t l_matr_size = size * size * sizeof(double);
    size_t vec_size = size * sizeof(double);
    hipMalloc((void**) &l_matr_dev, l_matr_size);
    hipMalloc((void**) &r_vec_dev, vec_size);
    hipMalloc((void**) &ret_vec_dev, vec_size);
    // Зануляем правый вектор
    fillVecKernel<<<N_blocks1, N_threads1>>>(ret_vec_dev, size, 0);
    // Копируем туда данные
    hipMemcpy(l_matr_dev, l_matr.get_arr(), l_matr_size, hipMemcpyHostToDevice);
    hipMemcpy(r_vec_dev, r_vec, vec_size, hipMemcpyHostToDevice);
    // Прямой ход
    // Приводим матрицу к верхнетреугольной
    double* mcoeff;
    hipMalloc((void**)&mcoeff, sizeof(double));
    for (size_t i = 0; i < size; i++) {
        for (size_t j = 0; j < i; j++) {
            calculateMCoeffKernel<<<N_threads_once, N_blocks_once >>>(l_matr_dev, size, i, j, mcoeff);
            plusRowKernel<<<N_blocks1, N_threads1>>>(l_matr_dev, r_vec_dev, size, i, j, mcoeff);
        }
     }
    hipFree(mcoeff);
    hipDeviceSynchronize();
    // Выводим единицы на главной диагонали
    // Вытаскиваем диагональные элементы, чтобы модифицировать матрицу
    double* diag_vec_dev;
    hipMalloc((void**)&diag_vec_dev, vec_size);
    diagElemsToVecKernel <<<N_blocks1, N_threads1 >>> (l_matr_dev, diag_vec_dev, size);
    // Диагональный элементы строк в единицы
    for (size_t i = 0; i < size; i++) {
        setDiagToOneKernel <<<N_blocks1, N_threads1 >>> (l_matr_dev, r_vec_dev, diag_vec_dev, size, i);
    }
    // И правый вектор
    rvecDiagDivKernel<<<N_blocks1, N_threads1 >>>(r_vec_dev, diag_vec_dev, size);
    hipFree(diag_vec_dev);

    //// Обратный ход
    //for (size_t i = 0; i < size; i++) {
    //    //setDiagToOneKernel << <N_blocks1, N_threads1 >> > (l_matr_dev, r_vec_dev, diag_vec_dev, size, i);
    //    
    //}

    //// Лепим возвращаемый вектор
    //double* ret_vec = new double[size];
    //hipMemcpy(ret_vec, ret_vec_dev, vec_size, hipMemcpyDeviceToHost);
    //vector<double> ret_vect;
    //for (size_t i = 0; i < size; i++) {
    //    ret_vect.push_back(ret_vec[i]);
    //}
    //// Возвращаем
    delete[] r_vec;
    //delete[] ret_vec;
    //return ret_vect;

    // Обратный ход
    double* l_matr_host = new double[size*size];
    double* r_vec_host = new double[size];
    hipMemcpy(l_matr_host, l_matr_dev, l_matr_size, hipMemcpyDeviceToHost);
    hipMemcpy(r_vec_host, r_vec_dev, vec_size, hipMemcpyDeviceToHost);
    vector<double>& ret_vect = *(new vector<double>(size));
    for (size_t i = 0; i < size; i++) {
        double sum = 0;
        for (size_t j = 0; j < i; j++) {
            sum += l_matr_host[(size - i - 1)*size + (size - j - 1)] * ret_vect[size - j - 1];
        }
        ret_vect[size - i - 1] = r_vec_host[size - i - 1] - sum;
    }
    return ret_vect;
}
